#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloWorldKernel() {
  int blockId = blockIdx.x;
  int threadIdX = threadIdx.x;
  int threadIdY = threadIdx.y;
  printf(
    "Hello World from Thread (%d, %d) in Block %d!\n", 
    threadIdX, threadIdY, blockId
  );
}

int main() {
  int n, m, k;
  
  printf("Enter n (number of blocks, 1-32): ");
  scanf("%d", &n);
  printf("Enter m (block dimension x, 1-32): ");
  scanf("%d", &m);
  printf("Enter k (block dimension y, 1-32): ");
  scanf("%d", &k);
  
  if (n < 1 || n > 32 || m < 1 || m > 32 || k < 1 || k > 32) {
    printf("Error: All inputs must be between 1 and 32\n");
    return 1;
  }
  
  printf("Hello World from the host!\n");
  
  dim3 blockDim(m, k);
  dim3 gridDim(n, 1);
  
  helloWorldKernel<<<gridDim, blockDim>>>();
  
  hipDeviceSynchronize();
  
  return 0;
}