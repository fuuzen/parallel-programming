#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>

#define EPSILON 0.000001

#define CUDA_CHECK(call) \
  do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
      fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
        hipGetErrorString(err)); \
      exit(1); \
    } \
  } while (0)

__global__ void transposeGlobal(double *A, double *AT, int n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x < n && y < n) {
    AT[y * n + x] = A[x * n + y];
  }
}

__global__ void transposeShared(double *A, double *AT, int n, int tile_size) {
  __shared__ double tile[32][32];
  int x = blockIdx.x * tile_size + threadIdx.x;
  int y = blockIdx.y * tile_size + threadIdx.y;

  if (x < n && y < n) {
    tile[threadIdx.y][threadIdx.x] = A[y * n + x];
  }
  __syncthreads();

  if (x < n && y < n) {
    AT[x * n + y] = tile[threadIdx.x][threadIdx.y];
  }
}

void initializeMatrix(double *A, int n) {
  for (int i = 0; i < n * n; i++) {
    A[i] = (double)rand() / RAND_MAX;
  }
}

bool verifyTransposed(double *A, double *A_T, int n) {
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      if (i == j && abs(A[i * n + j] - A_T[i * n + j]) > EPSILON) {
        return false;
      } else if (abs(A[i * n + j] - A_T[j * n + i]) > EPSILON) {
        return false;
      } 
    }
  }
  return true;
}

int main() {
  int n, tile_size;
  char memory_mode;
  printf("Enter matrix size n (512-2048): ");
  scanf("%d", &n);
  printf("Enter thread block size (e.g., 16, 32): ");
  scanf("%d", &tile_size);
  printf("Memory mode (G for global, S for shared): ");
  scanf(" %c", &memory_mode);

  if (n < 512 || n > 2048) {
    printf("Error: n must be between 512 and 2048\n");
    return 1;
  }
  if (tile_size != 16 && tile_size != 32) {
    printf("Error: tile_size must be 16 or 32\n");
    return 1;
  }
  if (memory_mode != 'G' && memory_mode != 'S') {
    printf("Error: memory_mode must be G or S\n");
    return 1;
  }

  size_t size = n * n * sizeof(double);
  
  double *h_A = (double *)malloc(size);
  double *h_AT = (double *)malloc(size);
  
  initializeMatrix(h_A, n);

  double *d_A, *d_AT;
  CUDA_CHECK(hipMalloc(&d_A, size));
  CUDA_CHECK(hipMalloc(&d_AT, size));
  
  CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  dim3 blockDim(tile_size, tile_size);
  dim3 gridDim((n + tile_size - 1) / tile_size, (n + tile_size - 1) / tile_size);

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  if (memory_mode == 'G') {
    transposeGlobal<<<gridDim, blockDim>>>(d_A, d_AT, n);
  } else {
    transposeShared<<<gridDim, blockDim>>>(d_A, d_AT, n, tile_size);
  }
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

  CUDA_CHECK(hipMemcpy(h_AT, d_AT, size, hipMemcpyDeviceToHost));

  if (verifyTransposed(h_A, d_AT, n)) {
    printf("Transposed correctly. GPU time: %.3f ms\n", milliseconds);
  } else {
    printf("Transpose result wrong. GPU time: %.3f ms\n", milliseconds);
  }

  free(h_A);
  free(h_AT);
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_AT));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}