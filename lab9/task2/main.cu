#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define EPSILON 0.000001

#define CUDA_CHECK(call) \
  do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
      fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
        hipGetErrorString(err)); \
      exit(1); \
    } \
  } while (0)

__global__ void transposeGlobal(double *A, double *AT, int n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x < n && y < n) {
    AT[y * n + x] = A[x * n + y];
  }
}

__global__ void transposeShared(double *A, double *AT, int n, int tile_size) {
  __shared__ double tile[32][32];
  int x = blockIdx.x * tile_size + threadIdx.x;
  int y = blockIdx.y * tile_size + threadIdx.y;

  if (x < n && y < n) {
    tile[threadIdx.y][threadIdx.x] = A[y * n + x];
  }
  __syncthreads();

  if (x < n && y < n) {
    AT[x * n + y] = tile[threadIdx.x][threadIdx.y];
  }
}

void initializeMatrix(double *A, int n) {
  for (int i = 0; i < n * n; i++) {
    A[i] = (double)rand() / RAND_MAX;
  }
}

int verifyTransposed(double *A, double *AT, int n) {
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      if (i == j && abs(A[i * n + j] - AT[i * n + j]) > EPSILON) {
        return 1;
      } else if (abs(A[i * n + j] - AT[j * n + i]) > EPSILON) {
        return 1;
      } 
    }
  }
  return 0;
}

int main( int argc, char *argv[] ) {
  int n = 2048;
  int tile_size = 32;
  char memory_mode = 'G';

  n = atoi(argv[1]);
  tile_size = atoi(argv[2]);
  memory_mode = argv[3][0];

  // if (n < 512 || n > 2048) {
  //   printf("Error: n must be between 512 and 2048\n");
  //   return 1;
  // }
  if (tile_size > 32) {
    printf("Error: tile_size must below 32\n");
    return 1;
  }
  if (memory_mode != 'G' && memory_mode != 'S') {
    printf("Error: memory_mode must be G or S\n");
    return 1;
  }

  size_t size = n * n * sizeof(double);
  
  double *h_A = (double *)malloc(size);
  double *h_AT = (double *)malloc(size);
  
  initializeMatrix(h_A, n);

  double *d_A, *d_AT;
  CUDA_CHECK(hipMalloc(&d_A, size));
  CUDA_CHECK(hipMalloc(&d_AT, size));
  
  CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  dim3 blockDim(tile_size, tile_size);
  dim3 gridDim((n + tile_size - 1) / tile_size, (n + tile_size - 1) / tile_size);

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  if (memory_mode == 'G') {
    transposeGlobal<<<gridDim, blockDim>>>(d_A, d_AT, n);
  } else {
    transposeShared<<<gridDim, blockDim>>>(d_A, d_AT, n, tile_size);
  }
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

  CUDA_CHECK(hipMemcpy(h_AT, d_AT, size, hipMemcpyDeviceToHost));

  printf("%.5E\n", milliseconds);

  int ret = verifyTransposed(h_A, h_AT, n);

  free(h_A);
  free(h_AT);
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_AT));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return ret;
}