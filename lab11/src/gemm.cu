#include "hip/hip_runtime.h"
#include "main.h"

__global__ void gemmKernel(
  float* __restrict__ A,
  float* __restrict__ B,
  float* __restrict__ C,
  sMatrixSize matrix_size
) {
  __shared__ float shareA[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float shareB[BLOCK_SIZE][BLOCK_SIZE];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by * BLOCK_SIZE + ty;
  int col = bx * BLOCK_SIZE + tx;
  float temp = 0;
  for(int i = 0; i < matrix_size.uiWA/BLOCK_SIZE; ++i){
    shareA[ty][tx] = A[row*matrix_size.uiWA + (i*BLOCK_SIZE + tx)];
    shareB[ty][tx] = B[(i*BLOCK_SIZE + ty)*matrix_size.uiWB + col];
    __syncthreads();
    for(int k = 0; k < BLOCK_SIZE; ++k){
      temp += shareA[ty][k] * shareB[k][tx];
      __syncthreads();
    }
  }
  C[row*matrix_size.uiWC + col] = temp;
}

// Tiling 版本的矩阵乘法，需要矩阵规模为 BLOCK_SIZE 的倍数
void gemm(float* A, float* B, float* C, sMatrixSize &matrix_size) {
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim(
    matrix_size.uiWB / blockDim.x,
    matrix_size.uiHA / blockDim.y
  );

  gemmKernel<<<gridDim, blockDim>>>(A, B, C, matrix_size);

  checkCudaErrors(hipDeviceSynchronize());
}