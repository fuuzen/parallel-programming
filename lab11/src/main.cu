#include "main.h"
#include "conv_sliding_window.h"
#include "conv_im2col_gemm.h"
#include "conv_cudnn.h"


void initialize(
  int argc,
  char **argv,
  int &devID,
  int &method,
  int &stride,
  char **input,
  char **output
) {
  // By default, we use device 0, otherwise we override the device ID based on
  // what is provided at the command line
  devID = 0;
  devID = findCudaDevice(argc, (const char **)argv);

  if (checkCmdLineFlag(argc, (const char **)argv, "method")) {
    // method 参数选择不同实现方法: 0 代表滑动窗口实现, 1 代表 im2col+gemm 实现, 其他代表 CuDNN 实现
    method = getCmdLineArgumentInt(argc, (const char **)argv, "method");
  }
  if (checkCmdLineFlag(argc, (const char **)argv, "input")) {
    getCmdLineArgumentString(argc, (const char **)argv, "input", input);
  }
  if (checkCmdLineFlag(argc, (const char **)argv, "output")) {
    getCmdLineArgumentString(argc, (const char **)argv, "output", output);
  }
  if (checkCmdLineFlag(argc, (const char **)argv, "stride")) {
    stride = getCmdLineArgumentInt(argc, (const char **)argv, "stride");
  }
}


void convolution(int devID, int method, int stride, char *input_file, char *output_file) {
  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
  int nIter = 30;
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  cv::Mat input, output;
  input = load_image(input_file);

  printf("Computing result using %s\n\n",
    method==0 ? "sliding window" :
    method==1 ? "im2col+gemm" :
    "CuDNN"
  );

  printf("Perform warm up convolution...");
  switch(method){
    case 0:
      output = conv_sliding_window(input, (float*)KERNEL, KERNEL_H, KERNEL_W, CHANNELS, stride);
      break;
    case 1:
      output = conv_im2col_gemm(input, (float*)KERNEL, KERNEL_H, KERNEL_W, CHANNELS, stride);
      break;
    default:
      output = conv_cudnn(input, (float*)KERNEL, KERNEL_H, KERNEL_W, CHANNELS, stride);
  }
  checkCudaErrors(hipGetLastError());
  save_image(output_file, output);
  printf("done.\n\n");
  printf("Output image has been saved to %s.\n\n", output_file);

  // Record the start event
  checkCudaErrors(hipEventRecord(start, NULL));

  // Evaluate performance
  printf("Evaluating performance...");
  for (int j = 0; j < nIter; j++) {
    switch(method){
      case 0:
        conv_sliding_window(input, (float*)KERNEL, KERNEL_H, KERNEL_W, CHANNELS, stride);
        break;
      case 1:
        conv_im2col_gemm(input, (float*)KERNEL, KERNEL_H, KERNEL_W, CHANNELS, stride);
        break;
      default:
        conv_cudnn(input, (float*)KERNEL, KERNEL_H, KERNEL_W, CHANNELS, stride);
    }
  }
  printf("done.\n\n");

  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, NULL));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  float msecPerConv = msecTotal / nIter;
  double flopsPerConv = 2.0 * CHANNELS * CHANNELS * output.rows * output.cols * KERNEL_H * KERNEL_W;
  double gigaFlops =
      (flopsPerConv * 1.0e-9f) / (msecPerConv / 1000.0f);
  printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n\n",
          gigaFlops, msecPerConv, flopsPerConv);
}

// Program main
int main(int argc, char **argv) {
  printf("[Convolution] - Starting...\n\n");

  int devID = 0, method = 2, stride = 1;
  char default_input[10] = "input.jpg";
  char default_output[11] = "output.jpg";
  char * input = default_input, * output = default_output;
  
  initialize(argc, argv, devID, method, stride, &input, &output);

  convolution(devID, method, stride, input, output);

  return 0;
}
