#include "hip/hip_runtime.h"
#include "conv_im2col_gemm.h"
#include "hip/hip_runtime_api.h"


__global__ void im2col_kernel(
  const float* input_data, float* col_data,
  int channels,
  int in_h, int in_w,
  int ker_h, int ker_w,
  int stride_h, int stride_w,
  int pad_h, int pad_w,
  int out_h, int out_w
) {
  int out_y = blockIdx.y * blockDim.y + threadIdx.y;
  int out_x = blockIdx.x * blockDim.x + threadIdx.x;

  if (out_y < out_h && out_x < out_w) {
    int col_index = out_y * out_w + out_x;
    for (int c = 0; c < channels; ++c) {
      for (int kh = 0; kh < ker_h; ++kh) {
        for (int kw = 0; kw < ker_w; ++kw) {
          int in_y = out_y * stride_h - pad_h + kh;
          int in_x = out_x * stride_w - pad_w + kw;

          int row_index = c * ker_h * ker_w + kh * ker_w + kw;
          int col_buffer_index = row_index * (out_h * out_w) + col_index;

          if (in_y >= 0 && in_y < in_h && in_x >= 0 && in_x < in_w) {
            col_data[col_buffer_index] = input_data[c * in_h * in_w + in_y * in_w + in_x];
          } else {
            col_data[col_buffer_index] = 0.0f;
          }
        }
      }
    }
  }
}


__global__ void gemm_kernel(
  const float* A,
  const float* B,
  float* C,
  int m, int n, int k
) {
  __shared__ float shareA[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float shareB[BLOCK_SIZE][BLOCK_SIZE];
  
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by * BLOCK_SIZE + ty;
  int col = bx * BLOCK_SIZE + tx;
  float temp = 0;
  
  for (int t = 0; t < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; t++) {
    if (row < m && t * BLOCK_SIZE + tx < n) {
      shareA[ty][tx] = A[row * n + t * BLOCK_SIZE + tx];
    } else {
      shareA[ty][tx] = 0.0f;
    }
    if (t * BLOCK_SIZE + ty < n && col < k) {
      shareB[ty][tx] = B[(t * BLOCK_SIZE + ty) * k + col];
    } else {
      shareB[ty][tx] = 0.0f;
    }
    __syncthreads();
    if (row < m && col < k) {
      for (int i = 0; i < BLOCK_SIZE; i++) {
        temp += shareA[ty][i] * shareB[i][tx];
      }
    }
    __syncthreads();
  }
  if (row < m && col < k) {
    C[row * k + col] = temp;
  }
}


cv::Mat conv_im2col_gemm(
  cv::Mat &image,
  float * kernel,
  int kernel_h,
  int kernel_w,
  int in_channels,
  int stride,
  int padding,
  int batch_size
) {
  int out_batch_size = batch_size;
  if (out_batch_size != 1) {
    std::cerr << "Sorry, only supports batch size = 1 now!" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  int out_channels = in_channels;
  int in_height = image.rows;
  int in_width = image.cols;
  int out_height = (in_height - kernel_h + 2 * padding) / stride + 1;
  int out_width = (in_width - kernel_w + 2 * padding) / stride + 1;
  if (out_height <= 0 || out_width <= 0) {
    std::cerr << "Invalid image size, kernel size, padding or stride!" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  int m = out_channels;
  int n = in_channels * kernel_h * kernel_w;
  int k = out_height * out_width;

  std::vector<cv::Mat> out_channels_mat;

  size_t kernel_bytes = out_channels * in_channels * kernel_h * kernel_w * sizeof(float);
  size_t input_bytes = batch_size * in_height * in_width * in_channels * sizeof(float);
  size_t output_bytes = m * k * sizeof(float);
  size_t col_bytes = n * k * sizeof(float);

  float *d_input, *d_kernel, *d_output, *d_col;
  hipMalloc(&d_input, input_bytes);
  hipMalloc(&d_col, col_bytes);
  hipMemcpy(d_input, image.ptr<float>(0), input_bytes, hipMemcpyHostToDevice);
  
  // im2col
  dim3 im2col_block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 im2col_grid(
    (out_width + im2col_block.x - 1) / im2col_block.x, 
    (out_height + im2col_block.y - 1) / im2col_block.y
  );
  im2col_kernel<<<im2col_grid, im2col_block>>>(
    d_input, d_col,
    in_channels,
    in_height, in_width,
    kernel_h, kernel_w,
    stride, stride,
    padding, padding,
    out_height, out_width
  );
  checkCudaErrors(hipDeviceSynchronize());

  hipFree(d_input);
  hipMalloc(&d_kernel, kernel_bytes);
  hipMalloc(&d_output, output_bytes);
  hipMemcpy(d_kernel, kernel, kernel_bytes, hipMemcpyHostToDevice);
  
  // gemm
  dim3 gemm_block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gemm_grid(
    (k + gemm_block.x - 1) / gemm_block.x,
    (m + gemm_block.y - 1) / gemm_block.y
  );
  gemm_kernel<<<gemm_grid, gemm_block>>>(
    d_kernel,
    d_col,
    d_output,
    m, n, k
  );
  
  cv::Mat output_image(out_height, out_width, CV_32FC3);
  checkCudaErrors(hipDeviceSynchronize());
  hipMemcpy(output_image.ptr(0), d_output, output_bytes, hipMemcpyDeviceToHost);

  hipFree(d_col);
  hipFree(d_kernel);
  hipFree(d_output);
  return output_image;
}