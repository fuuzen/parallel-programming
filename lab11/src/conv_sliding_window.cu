#include "hip/hip_runtime.h"
#include "conv_sliding_window.h"


__global__ void conv2d_kernel(
  const float* input, const float* kernel, float* output,
  int input_h, int input_w,
  int kernel_h, int kernel_w,
  int channels, int stride, int padding,
  int output_h, int output_w
) {
  int out_x = blockIdx.x * blockDim.x + threadIdx.x;
  int out_y = blockIdx.y * blockDim.y + threadIdx.y;
  if (out_x < output_w && out_y < output_h) {
    float acc = 0.0f;
    int in_base_y = out_y * stride - padding;
    int in_base_x = out_x * stride - padding;
    for (int c = 0; c < channels; ++c) {
      for (int kh = 0; kh < kernel_h; ++kh) {
        for (int kw = 0; kw < kernel_w; ++kw) {
          int in_y = in_base_y + kh;
          int in_x = in_base_x + kw;
          if (in_y >= 0 && in_y < input_h && in_x >= 0 && in_x < input_w) {
            int input_idx = (in_y * input_w + in_x) * channels + c;
            int kernel_idx = (kh * kernel_w + kw) * channels + c;
            acc += input[input_idx] * kernel[kernel_idx];
          }
        }
      }
    }
    output[out_y * output_w + out_x] = acc;
  }
}


cv::Mat conv_sliding_window(
  cv::Mat &image,
  float * kernel,
  int kernel_h,
  int kernel_w,
  int in_channels,
  int stride,
  int padding,
  int batch_size
) {
  int out_batch_size = batch_size;
  if (batch_size != 1) {
    std::cerr << "Sorry, only supports batch size = 1 now!" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  int out_channels = in_channels;
  int in_height = image.rows;
  int in_width = image.cols;
  int out_height = (in_height - kernel_h + 2 * padding) / stride + 1;
  int out_width = (in_width - kernel_w + 2 * padding) / stride + 1;
  if (out_height <= 0 || out_width <= 0) {
    std::cerr << "Invalid image size, kernel size, padding or stride!" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  size_t kernel_bytes = in_channels * kernel_h * kernel_w * sizeof(float);  // only 1 out channel
  size_t input_bytes = batch_size * in_height * in_width * in_channels * sizeof(float);
  size_t output_bytes = out_batch_size * out_height * out_width * sizeof(float);  // only 1 out channel

  std::vector<cv::Mat> mv;

  float *d_input, *d_kernel, *d_output;
  hipMalloc(&d_input, input_bytes);
  hipMalloc(&d_kernel, kernel_bytes);
  hipMalloc(&d_output, output_bytes);
  hipMemcpy(d_input, image.ptr<float>(0), input_bytes, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(
    (out_width + threadsPerBlock.x - 1) / threadsPerBlock.x, 
    (out_height + threadsPerBlock.y - 1) / threadsPerBlock.y
  );

  for (int i = 0; i < out_channels; ++i) {
    hipMemcpy(d_kernel, kernel + (kernel_bytes/sizeof(float)), kernel_bytes, hipMemcpyHostToDevice);
    
    conv2d_kernel<<<numBlocks, threadsPerBlock>>>(
      d_input, d_kernel, d_output,
      in_height, in_width,
      kernel_h, kernel_w,
      in_channels, stride, padding,
      out_height, out_width
    );
    
    mv.emplace_back(cv::Mat(out_height, out_width, CV_32FC1));
    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(mv[i].ptr(0), d_output, output_bytes, hipMemcpyDeviceToHost);
  }

  cv::Mat output_image(out_height, out_width, CV_32FC3);
  cv::merge(mv, output_image);

  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  return output_image;
}