#include "conv_cudnn.h"


cv::Mat conv_cudnn(
  cv::Mat &image,
  float * kernel,
  int kernelH,
  int kernelW,
  int in_channels,
  int stride,
  int padding,
  int batch_size
) {
  hipdnnHandle_t cudnn;
  checkCUDNN(hipdnnCreate(&cudnn));

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
    input_descriptor,
    /*format=*/HIPDNN_TENSOR_NHWC,
    /*dataType=*/HIPDNN_DATA_FLOAT,
    /*batch_size=*/batch_size,
    /*channels=*/in_channels,
    /*image_height=*/image.rows,
    /*image_width=*/image.cols
  ));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
    /*dataType=*/HIPDNN_DATA_FLOAT,
    /*format=*/HIPDNN_TENSOR_NCHW,
    /*out_channels=*/in_channels,
    /*in_channels=*/in_channels,
    /*kernel_height=*/kernelH,
    /*kernel_width=*/kernelW
  ));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
    /*pad_height=*/padding,
    /*pad_width=*/padding,
    /*vertical_stride=*/stride,
    /*horizontal_stride=*/stride,
    /*dilation_height=*/1,
    /*dilation_width=*/1,
    /*mode=*/HIPDNN_CROSS_CORRELATION,
    /*computeType=*/HIPDNN_DATA_FLOAT
  ));

  int out_batch_size, out_channels, out_height, out_width;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
    convolution_descriptor,
    input_descriptor,
    kernel_descriptor,
    &out_batch_size,
    &out_channels,
    &out_height,
    &out_width
  ));
  
  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
    /*format=*/HIPDNN_TENSOR_NHWC,
    /*dataType=*/HIPDNN_DATA_FLOAT,
    /*batch_size=*/out_batch_size,
    /*channels=*/out_channels,
    /*image_height=*/out_height,
    /*image_width=*/out_width
  ));

  hipdnnConvolutionFwdAlgoPerf_t algo_perf;
  int returnedAlgoCount = 0;
  checkCUDNN(cudnnGetConvolutionForwardAlgorithm_v7(
    cudnn,
    input_descriptor,
    kernel_descriptor,
    convolution_descriptor,
    output_descriptor,
    1,
    &returnedAlgoCount,
    &algo_perf
  ));
  hipdnnConvolutionFwdAlgo_t convolution_algorithm = algo_perf.algo;

  size_t workspace_bytes = 0;
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnn,
    input_descriptor,
    kernel_descriptor,
    convolution_descriptor,
    output_descriptor,
    convolution_algorithm,
    &workspace_bytes
  ));

  size_t kernel_bytes = sizeof(float) * in_channels * out_channels * kernelH * kernelW;
  size_t input_bytes = batch_size * image.rows * image.cols * in_channels * sizeof(float);
  size_t output_bytes = out_batch_size * out_channels * out_height * out_width * sizeof(float);

  float *d_input, *d_kernel, *d_output, *d_workspace;
  hipMalloc(&d_workspace, workspace_bytes);
  hipMalloc(&d_input, input_bytes);
  hipMalloc(&d_kernel, kernel_bytes);
  hipMalloc(&d_output, output_bytes);
  hipMemset(d_output, 0, output_bytes);
  hipMemcpy(d_input, image.ptr<float>(0), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel, kernel_bytes, hipMemcpyHostToDevice);

  const float alpha = 1, beta = 0;
  checkCUDNN(hipdnnConvolutionForward(
    cudnn,
    &alpha,
    input_descriptor,
    d_input,
    kernel_descriptor,
    d_kernel,
    convolution_descriptor,
    convolution_algorithm,
    d_workspace,
    workspace_bytes,
    &beta,
    output_descriptor,
    d_output
  ));

  cv::Mat output_image(out_height, out_width, CV_32FC3);
  hipMemcpy(output_image.ptr(0), d_output, output_bytes, hipMemcpyDeviceToHost);

  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroy(cudnn);
  return output_image;
}